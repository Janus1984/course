#include <cstdio>
#include <hip/hip_runtime.h>

__global__ void kernel() {

    // xyz三个维度的设计初衷是为了适配图像的三维，但其实和只使用一维是一样的。

    printf("Block (%d,%d,%d) of (%d,%d,%d), Thread (%d,%d,%d) of (%d,%d,%d)\n",
           blockIdx.x, blockIdx.y, blockIdx.z,
           gridDim.x, gridDim.y, gridDim.z,
           threadIdx.x, threadIdx.y, threadIdx.z,
           blockDim.x, blockDim.y, blockDim.z);
}

int main() {
    kernel<<<dim3(2, 1, 1), dim3(2, 2, 2) >>>();
    hipDeviceSynchronize();
    return 0;
}
