#include <cstdio>
#include <hip/hip_runtime.h>

__global__ void kernel() {
    printf("Block (%d,%d) of (%d,%d), Thread (%d,%d) of (%d,%d)\n",
           blockIdx.x, blockIdx.y,
           gridDim.x, gridDim.y,
           threadIdx.x, threadIdx.y,
           blockDim.x, blockDim.y);
}

int main() {
    // 二维的话只要将最后一个维度设为1就行了
    kernel<<<dim3(2, 1, 1), dim3(3, 2, 1)>>>();
    hipDeviceSynchronize();
    return 0;
}
