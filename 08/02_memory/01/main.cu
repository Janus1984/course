#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>

// __global__ 函数不能有返回值，因为gpu是异步的，如下main函数中获取ret时，还没有执行hipDeviceSynchronize,所以会报错。
__global__ int kernel() {
    return 42;
}

int main() {
    int ret = kernel<<<1, 1>>>();
    hipDeviceSynchronize();
    printf("%d\n", ret);
    return 0;
}
