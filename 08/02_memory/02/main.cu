#include <cstdio>
#include <hip/hip_runtime.h>

// 既然不能返回，那作为指针传入局部变量的引用可行么？ 结果是ret依然等于0。
__global__ void kernel(int *pret) {
    *pret = 42;
}

int main() {
    int ret = 0;
    kernel<<<1, 1>>>(&ret);
    hipDeviceSynchronize();
    printf("%d\n", ret);
    return 0;
}
