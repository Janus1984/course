#include <cstdio>
#include <hip/hip_runtime.h>

__global__ void kernel(int *pret) {
    *pret = 42;
}

int main() {
    int ret = 0;
    kernel<<<1, 1>>>(&ret);
    // cuda的函数出错时，不会立即终止程序，也不会抛出C++异常，而是返回一个错误代码，类型是cudaError_t，其实就是个enum类型，相当于int
    hipError_t err = hipDeviceSynchronize();
    printf("error code: %d\n", err);  // 700
    printf("error name: %s\n", hipGetErrorName(err));  // cudaErrorIllegalAddress, 意思是访问了非法地址，和CPU上的段错误差不多。
    printf("%d\n", ret);
    return 0;
}
