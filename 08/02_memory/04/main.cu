#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"

__global__ void kernel(int *pret) {
    *pret = 42;
}

int main() {
    int ret = 0;
    kernel<<<1, 1>>>(&ret);

    // CUDA toolkit提供了非常有用的工具，比如 hip/hip_runtime_api.h，它定义了 checkCudaErrors,会自动检查错误代码并打印在终端，然后退出。
    checkCudaErrors(hipDeviceSynchronize());
    return 0;
}
