#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"

__global__ void kernel(int *pret) {
    *pret = 42;
}

int main() {
    // 回到报错原因，为什么会报非法地址，那试着分配一块内存试试？ 依然报错，因为malloc是在cpu上分配的内存。
    int *pret = (int *)malloc(sizeof(int));
    kernel<<<1, 1>>>(pret);
    checkCudaErrors(hipDeviceSynchronize());
    free(pret);
    return 0;
}
