#include <cstdio>
#include <hip/hip_runtime.h>

__device__ void say_hello()
{
    printf("Hello,world form GPU!\n");
}

__host__ void say_hello_host()
{
    printf("Hello, world form CPU!\n");
}

__global__ void kernel()
{
    say_hello();
}

int main()
{
    kernel<<<1,1>>>();
    hipDeviceSynchronize();
    say_hello_host();
    return 0;
}