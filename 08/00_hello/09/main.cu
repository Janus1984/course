#include <cstdio>
#include <hip/hip_runtime.h>

__host__ __device__ void say_hello() {

// CUDA编译器具有多段编译的特点，会先将__host__的函数提取出来交给gcc编译生成CPU的指令码，
// 然后将__global__ __device__函数提取出来交给nvcc编译生成GPU的指令码，
// 然后将指令码链接到kernel函数
// 可以用__CUDA_ARCH__宏来实现两份代码，达到类似重载的效果

#ifdef __CUDA_ARCH__
    printf("Hello, world from GPU!\n");
#else
    printf("Hello, world from CPU!\n");
#endif
}

__global__ void kernel() {
    say_hello();
}

int main() {
    kernel<<<1, 1>>>();
    hipDeviceSynchronize();
    say_hello();
    return 0;
}
