#include <cstdio>
#include <hip/hip_runtime.h>

// cuda会把constexpr函数自动变成修饰__host__,__device__
// 不过必须指定 --expt-relaxed-constexpr 这个选项才能用这个特性
constexpr const char *cuthead(const char *p) {
    return p + 1;
}

__global__ void kernel() {
    printf(cuthead("Gello, world!\n"));
}

int main() {
    kernel<<<1, 1>>>();
    hipDeviceSynchronize();
    printf(cuthead("Cello, world!\n"));
    return 0;
}
