#include <cstdio>
#include <hip/hip_runtime.h>

__host__ __device__ void say_hello() {

// __CUDA_ARCH__ 其实是个版本号，表示GPU架构版本，编译器默认的是最老的52版本，能兼容GTX900以上显卡
#ifdef __CUDA_ARCH__
    printf("Hello, world from GPU architecture %d!\n", __CUDA_ARCH__);
#else
    printf("Hello, world from CPU!\n");
#endif
}

__global__ void kernel() {
    say_hello();
}

int main() {
    kernel<<<1, 1>>>();
    hipDeviceSynchronize();
    say_hello();
    return 0;
}
