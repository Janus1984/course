
#include <hip/hip_runtime.h>
#include <cstdio>

// __global__ 用于核函数，性质有点类似于cpu上的main函数
__global__ void kernel() {
    printf("Hello, world!\n");
}

int main() {
    kernel<<<1, 1>>>();
    return 0;
}
