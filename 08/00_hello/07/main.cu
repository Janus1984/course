#include <cstdio>
#include <hip/hip_runtime.h>

// __host__ 和 __device__ 可以一起用，也就是在host和device上都可以调用
__host__ __device__ void say_hello() {
    printf("Hello, world!\n");
}

__global__ void kernel() {
    say_hello();
}

int main() {
    kernel<<<1, 1>>>();
    hipDeviceSynchronize();
    say_hello();
    return 0;
}
