#include <cstdio>
#include <hip/hip_runtime.h>

__global__ void kernel() {
    printf("Hello, world!\n");
}

int main() {
    kernel<<<1, 1>>>();
    // CPU和GPU之间的通信是异步的，调用kernel函数后只是把这个任务推送到GPU的执行队列上，然后立即返回，并不会等待执行完毕
    // 可以调用cudaDeviceSynchronize(),让cpu陷入等待，等GPU完成队列的所有任务后再返回
    hipDeviceSynchronize();
    return 0;
}
